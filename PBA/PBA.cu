
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include<string.h>
#include<time.h>

/*
 * Brandon Cercone
 * Parallel Banker's Algorithm
 */
#define THREADS_PER_BLOCK 1024
#define NUM_OF_BLOCKS 16
__global__ void r_check(int *avail, int *alloc, int *need, int *out, int p, int *flag){
    //__shared__ int s_avail[THREADS_PER_BLOCK];
    //int bound = r * p;
    /* Index into the allocation vector using the thread id */
    int t_index = threadIdx.x;
    /* Index into the need & alloc matrix */
    int index = threadIdx.x + blockIdx.x * blockDim.x + (p * blockDim.x);
    //s_avail[t_index] = avail[t_index];
    //__syncthreads();
    //out[index] = (avail[t_index] >= need[index]) ? 1 : 0;
    //int t_need = need[index];
    //int t_avail = avail[t_index];
    //atomicOr(flag, t_avail - t_need);
    atomicOr(flag, avail[t_index] - need[index]);
    //if (t_avail < t_need) *flag = 0;
    //if (avail[t_index] < need[index]) *flag = 0;
    //do {
    //    //out[index] = (s_avail[t_index] >= need[index]) ? 1 : 0;
    //    out[index] = (avail[t_index] >= need[index]) ? 1 : 0;
    //    index += (NUM_OF_BLOCKS * r); 
    //} while (index < bound);
}
__global__ void m_r_check(int *avail, int *need, int *flag){
    int t_index = threadIdx.x;
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    atomicOr(&flag[blockIdx.x], avail[t_index] - need[index]);
}
__global__ void add_r(int *avail, int *alloc, int *p){
    int t_index = threadIdx.x;
    int index = threadIdx.x + blockIdx.x * blockDim.x + (*p * blockDim.x);
    avail[t_index] += alloc[index];
}
__global__ void set_zero(int *out){
    out[threadIdx.x + blockIdx.x * blockDim.x] = 0;
}
__global__ void p_search(int *out,int *seen,int *proc){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (out[index] >= 0 && !seen[index]){
        *proc = 1;
        seen[index] = 1;
    }
}
void store_vector(int *array, char *line){
	char *tokens = strtok(line,",");
	int i = 0;
	while (tokens != NULL){
		array[i++] = atoi(tokens);
		tokens = strtok(NULL,",");
	}
}
int main (int argc, char *argv[]) {
    float eTime;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	int p,r,num_processes,num_resources,size;
	FILE *file;
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	char *tokens;
    clock_t t;
    int index = 0;
	if (argc < 2){
		printf("ERROR: args=%d",argc);
		exit(EXIT_FAILURE);
    }
	/* Arg 1: Filename for matrix and resource vector */
	file=fopen(argv[1], "r");
	if (file == NULL)
		exit(EXIT_FAILURE);
	/* First line in the file is the number of processes */
	read = getline(&line, &len, file);
	num_processes = atoi(line);
	/* Second line in the file is the number of resources */
	read = getline(&line, &len, file);
	num_resources = atoi(line);
	/* Allocate memory for resource vectors and matrix */
    size = num_processes * num_resources * sizeof(int);
	int r_avail[num_resources];
    //int r_out[num_processes] = {0};
    int *r_out;
	int p_sequence[num_processes];
	int p_seen[num_processes];
    int *p_next;
	int *r_alloc, *r_max, *r_need;
	int *gpu_r_avail, *gpu_r_alloc, *gpu_r_need, *gpu_r_out, *gpu_p_next, *gpu_r_seen;
    r_alloc = (int *)malloc(size);
    r_max = (int *)malloc(size);
    r_need = (int *)malloc(size);
	for (p = 0; p < num_processes; p++){
		p_seen[p] = 0;
		p_sequence[p] = -1;
	}
	/* Fourth line in file is the resources available vector */
	read = getline(&line, &len, file);
	store_vector(r_avail,line);
	/* Read in the maximum reource needed matrix from file */
	for (p = 0; p < num_processes; p++){
		read = getline(&line, &len, file);
		tokens = strtok(line,",");
		while (tokens != NULL){
			r_max[index++] = atoi(tokens);
			tokens = strtok(NULL,",");
		}
	}
	/* Read in the reource allocation matrix from file */
	/* Calculate values for the need matrix */
    index = 0;
	for (p = 0; p < num_processes; p++){
		read = getline(&line, &len, file);
		tokens = strtok(line,",");
		while (tokens != NULL){
			r_alloc[index] = atoi(tokens);
			r_need[index] = r_max[index] - r_alloc[index];
			tokens = strtok(NULL,",");
			index++;
		}
	}
	fclose(file);
	if (line)
		free(line);
    hipEventRecord(start,0);
    t = clock();
    /* Allocate memory on the GPU */
    hipMalloc((void **)&gpu_r_avail, num_resources * sizeof(int));
    hipMalloc((void **)&gpu_r_alloc, size);
    hipMalloc((void **)&gpu_r_need, size);
    hipMalloc((void **)&gpu_r_out, num_processes * sizeof(int));
    hipMalloc((void **)&gpu_r_seen, num_processes * sizeof(int));
    /* Copy alloc and need to GPU */
    hipMemcpy(gpu_r_avail, r_avail, num_resources * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_r_alloc, r_alloc, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_r_need, r_need, size, hipMemcpyHostToDevice);
    //set_zero<<<1,THREADS_PER_BLOCK>>>(gpu_r_out);
    //cudaMemcpy(r_out, gpu_r_out, num_processes * sizeof(int), cudaMemcpyDeviceToHost);
    /* Launch kernel */
    int flag,counter;
    int *gpu_flag;
    hipMalloc((void **)&gpu_flag, sizeof(int));
    hipMalloc((void **)&gpu_p_next, sizeof(int));
    hipHostMalloc((void **)&p_next, sizeof(int), hipHostMallocDefault);
    hipHostMalloc((void **)&r_out, num_processes * sizeof(int), hipHostMallocDefault);
    counter = 0;
    int blocks = num_processes / 100;
    set_zero<<<blocks,100>>>(gpu_r_seen);
    while(counter < num_processes){
        t = clock();
        set_zero<<<blocks,100>>>(gpu_r_out);
        t = clock() - t;
        printf("zero time: %f\n", ((double)t)/CLOCKS_PER_SEC);

        t = clock();
        m_r_check<<<num_processes,THREADS_PER_BLOCK>>>(gpu_r_avail,gpu_r_need,gpu_r_out);
        t = clock() - t;
        printf("m_r check time: %f\n", ((double)t)/CLOCKS_PER_SEC);

        t = clock();
        p_search<<<blocks,100>>>(gpu_r_out,gpu_r_seen,gpu_p_next);
        t = clock() - t;
        //printf("search time: %f\n", ((double)t)/CLOCKS_PER_SEC);

        t = clock();
        hipMemcpy(&p_next, gpu_p_next, sizeof(int), hipMemcpyDeviceToHost);
        //cudaMemcpy(r_out, gpu_r_out, num_processes * sizeof(int), cudaMemcpyDeviceToHost);
        t = clock() - t;
        printf("xfer time: %f\n", ((double)t)/CLOCKS_PER_SEC);
        printf("p_next: %d\n", *p_next);
        if (*p_next >= 0){
            p_sequence[counter++] = *p_next;
            add_r<<<1,THREADS_PER_BLOCK>>>(gpu_r_avail,gpu_r_alloc,p_next);
        }
        else{
                printf("DENIED\n");
                break;
        }

        //flag = 0;
        //for (p = 0; p < num_processes; p++){
        //    printf("%d\n", r_out[p]);
        //    if (r_out[p] >= 0 && !p_seen[p]){
        //        add_r<<<1,THREADS_PER_BLOCK>>>(gpu_r_avail,gpu_r_alloc,p);
        //        p_seen[p] = 1;
        //        p_sequence[counter++] = p;
        //        //printf("%d\n", p);
        //        flag = 1;
        //        break;
        //    }
        //    //r_out[p] = 0;
        //}
        //if (!flag){
        //        printf("DENIED");
        //        break;
        //}
        //for (p = 0; p < num_processes; p++)
        //    r_out[p] = 0;
    }
        //for (p = 0; p < num_processes; p++){
        //    if (p_seen[p]) continue;
        //    flag = 1;
        //    cudaMemcpy(gpu_flag,&flag,sizeof(int),cudaMemcpyHostToDevice);
        //    r_check<<<1,THREADS_PER_BLOCK>>>(gpu_r_avail, gpu_r_alloc, gpu_r_need, gpu_r_out, p, gpu_flag);
        //    cudaMemcpy(&flag,gpu_flag,sizeof(int),cudaMemcpyDeviceToHost);
        //    if (flag>=0){
        //        add_r<<<1,THREADS_PER_BLOCK>>>(gpu_r_avail,gpu_r_alloc,p);
        //        p_seen[p] = 1;
        //        p_sequence[counter] = p;
        //        counter++;
        //        break;
        //    }
        //}
        //if (!flag){
        //    printf("DENIED\n");
        //    break;
        //}
    //}
    //for (int k = 0; k < num_processes; k++){
    ////r_check<<<NUM_OF_BLOCKS,THREADS_PER_BLOCK>>>(gpu_r_avail, gpu_r_alloc, gpu_r_need, gpu_r_out, num_processes, num_resources);
    //r_check<<<1,THREADS_PER_BLOCK>>>(gpu_r_avail, gpu_r_alloc, gpu_r_need, gpu_r_out, k, num_resources);
    //}
    //printf("%s\n", cudaGetErrorString(cudaGetLastError()));
    /* Copy result to host */
    //cudaMemcpy(r_out, gpu_r_out,size, cudaMemcpyDeviceToHost);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&eTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //printf("%s\n", cudaGetErrorString(cudaGetLastError()));
    /* Free GPU memory */
    hipFree(gpu_r_avail);
    hipFree(gpu_r_alloc);
    hipFree(gpu_r_need);
    hipFree(gpu_r_out);
    hipFree(gpu_flag);
    hipHostFree(p_next);
    t = clock() - t;

    for (r = 0; r < num_processes; r++){
        printf("%d ", p_sequence[r]);
    }
    printf("\n\n");
    /*
	for (p = 0; p < num_processes; p++){
	    for (r = 0; r < num_resources; r++){
            printf("%d ", r_alloc[num_resources * p + r]);
        }
        printf("\n");
	}
    printf("\n\n");
	for (p = 0; p < num_processes; p++){
	    for (r = 0; r < num_resources; r++){
            printf("%d ", r_need[num_resources * p + r]);
        }
        printf("\n");
	}
    printf("\n");
	for (p = 0; p < num_processes; p++){
	    for (r = 0; r < num_resources; r++){
            printf("%d ", r_out[num_resources * p + r]);
        }
        printf("\n");
	}*/

    printf("%d %f\n", num_processes, eTime/1000.0);
    //printf("%d %f\n", num_processes, ((double)t)/CLOCKS_PER_SEC);

    free(r_alloc);
    free(r_max);
    free(r_need);
	exit(EXIT_SUCCESS);
}
