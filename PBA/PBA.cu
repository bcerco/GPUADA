
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include<string.h>

/*
 * Brandon Cercone
 * Parallel Banker's Algorithm
 */
#define THREADS_PER_BLOCK 1024
#define NUM_OF_BLOCKS 16
__global__ void r_check(int *avail, int *alloc, int *need, int *out, int p, int r){
    __shared__ int s_avail[THREADS_PER_BLOCK];
    int bound = r * p;
    /* Index into the allocation vector using the thread id */
    int t_index = threadIdx.x;
    /* Index into the need & alloc matrix */
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    s_avail[t_index] = avail[t_index];
    __syncthreads();
    do {
        out[index] = (s_avail[t_index] >= need[index]) ? 1 : 0;
        //out[index] = (avail[t_index] >= need[index]) ? 1 : 0;
        index += (NUM_OF_BLOCKS * r); 
    } while (index < bound);
}
void store_vector(int *array, char *line){
	char *tokens = strtok(line,",");
	int i = 0;
	while (tokens != NULL){
		array[i++] = atoi(tokens);
		tokens = strtok(NULL,",");
	}
}
int main (int argc, char *argv[]) {
    float eTime;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	int p,r,num_processes,num_resources,size;
	FILE *file;
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	char *tokens;
    int index = 0;
	if (argc < 2){
		printf("ERROR: args=%d",argc);
		exit(EXIT_FAILURE);
    }
	/* Arg 1: Filename for matrix and resource vector */
	file=fopen(argv[1], "r");
	if (file == NULL)
		exit(EXIT_FAILURE);
	/* First line in the file is the number of processes */
	read = getline(&line, &len, file);
	num_processes = atoi(line);
	/* Second line in the file is the number of resources */
	read = getline(&line, &len, file);
	num_resources = atoi(line);
	/* Allocate memory for resource vectors and matrix */
    size = num_processes * num_resources * sizeof(int);
	int r_avail[num_resources];
	int p_sequence[num_processes];
	int p_seen[num_processes];
	int *r_alloc, *r_max, *r_need, *r_out;
	int *gpu_r_avail, *gpu_r_alloc, *gpu_r_need, *gpu_r_out;
    r_alloc = (int *)malloc(size);
    r_max = (int *)malloc(size);
    r_need = (int *)malloc(size);
    r_out = (int *)malloc(size);
	for (p = 0; p < num_processes; p++){
		p_seen[p] = 0;
		p_sequence[p] = -1;
	}
	/* Fourth line in file is the resources available vector */
	read = getline(&line, &len, file);
	store_vector(r_avail,line);
	/* Read in the maximum reource needed matrix from file */
	for (p = 0; p < num_processes; p++){
		read = getline(&line, &len, file);
		tokens = strtok(line,",");
		while (tokens != NULL){
			r_max[index++] = atoi(tokens);
			tokens = strtok(NULL,",");
		}
	}
    printf("INDEX %d\n", index);
	/* Read in the reource allocation matrix from file */
	/* Calculate values for the need matrix */
    index = 0;
	for (p = 0; p < num_processes; p++){
		read = getline(&line, &len, file);
		tokens = strtok(line,",");
		while (tokens != NULL){
			r_alloc[index] = atoi(tokens);
			r_need[index] = r_max[index] - r_alloc[index];
			tokens = strtok(NULL,",");
			index++;
		}
	}
	fclose(file);
	if (line)
		free(line);
    hipEventRecord(start,0);
    /* Allocate memory on the GPU */
    hipMalloc((void **)&gpu_r_avail, num_resources * sizeof(int));
    hipMalloc((void **)&gpu_r_alloc, size);
    hipMalloc((void **)&gpu_r_need, size);
    hipMalloc((void **)&gpu_r_out, size);
    /* Copy alloc and need to GPU */
    hipMemcpy(gpu_r_avail, r_avail, num_resources * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_r_alloc, r_alloc, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_r_need, r_need, size, hipMemcpyHostToDevice);
    /* Launch kernel */
    r_check<<<NUM_OF_BLOCKS,THREADS_PER_BLOCK>>>(gpu_r_avail, gpu_r_alloc, gpu_r_need, gpu_r_out, num_processes, num_resources);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    /* Copy result to host */
    hipMemcpy(r_out, gpu_r_out,size, hipMemcpyDeviceToHost);
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&eTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    printf("%s\n", hipGetErrorString(hipGetLastError()));
    /* Free GPU memory */
    hipFree(gpu_r_avail);
    hipFree(gpu_r_alloc);
    hipFree(gpu_r_need);
    hipFree(gpu_r_out);

    /*for (r = 0; r < num_resources; r++){
        printf("%d ", r_avail[r]);
    }
    printf("\n\n");
	for (p = 0; p < num_processes; p++){
	    for (r = 0; r < num_resources; r++){
            printf("%d ", r_alloc[num_resources * p + r]);
        }
        printf("\n");
	}
    printf("\n\n");
	for (p = 0; p < num_processes; p++){
	    for (r = 0; r < num_resources; r++){
            printf("%d ", r_need[num_resources * p + r]);
        }
        printf("\n");
	}
    printf("\n");
	for (p = 0; p < num_processes; p++){
	    for (r = 0; r < num_resources; r++){
            printf("%d ", r_out[num_resources * p + r]);
        }
        printf("\n");
	}*/

    printf("GPU Time: %f seconds\n", eTime/1000.0);

    free(r_alloc);
    free(r_max);
    free(r_need);
	exit(EXIT_SUCCESS);
}
