
#include <hip/hip_runtime.h>
#include<stdio.h>
#include<stdlib.h>
#include<malloc.h>
#include<string.h>
#include<time.h>

/*
 * Brandon Cercone
 * Parallel Banker's Algorithm
 */
#define THREADS_PER_BLOCK 1024
#define NUM_OF_BLOCKS 16
__global__ void m_r_check(int *avail, int *need, int *flag, int *map){
    int t_index = threadIdx.x;
    int b_index = map[blockIdx.x];
    //int index = threadIdx.x + blockIdx.x * blockDim.x;
    //atomicOr(&flag[blockIdx.x], avail[t_index] - need[index]);
    int index = threadIdx.x + b_index * blockDim.x;
    atomicOr(&flag[b_index], avail[t_index] - need[index]);
}
__global__ void add_r(int *avail, int *alloc, int p){
    int t_index = threadIdx.x;
    int index = threadIdx.x + blockIdx.x * blockDim.x + (p * blockDim.x);
    avail[t_index] += alloc[index];
}
__global__ void set_zero(int *out){
    out[threadIdx.x + blockIdx.x * blockDim.x] = 0;
}
__global__ void init_map(int *map){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    map[index] = index;
}
__global__ void update_map(int *map, int p){
    int index = threadIdx.x + blockIdx.x * blockDim.x;
    if (map[index] >= p){
        map[index]++;
    }
}
void store_vector(int *array, char *line){
	char *tokens = strtok(line,",");
	int i = 0;
	while (tokens != NULL){
		array[i++] = atoi(tokens);
		tokens = strtok(NULL,",");
	}
}
int main (int argc, char *argv[]) {
    float eTime;
    hipEvent_t start,stop;
    hipEventCreate(&start);
    hipEventCreate(&stop);
	int p,r,num_processes,num_resources,size;
	FILE *file;
	char * line = NULL;
	size_t len = 0;
	ssize_t read;
	char *tokens;
    clock_t t;
    int index = 0;
	if (argc < 2){
		printf("ERROR: args=%d",argc);
		exit(EXIT_FAILURE);
    }
	/* Arg 1: Filename for matrix and resource vector */
	file=fopen(argv[1], "r");
	if (file == NULL)
		exit(EXIT_FAILURE);
	/* First line in the file is the number of processes */
	read = getline(&line, &len, file);
	num_processes = atoi(line);
	/* Second line in the file is the number of resources */
	read = getline(&line, &len, file);
	num_resources = atoi(line);
	/* Allocate memory for resource vectors and matrix */
    size = num_processes * num_resources * sizeof(int);
	int r_avail[num_resources];
    //int r_out[num_processes] = {0};
    int *r_out;
	int p_sequence[num_processes];
	int p_seen[num_processes];
	int *r_alloc, *r_max, *r_need;
	int *gpu_r_avail, *gpu_r_alloc, *gpu_r_need, *gpu_r_out, *gpu_b_map;
    r_alloc = (int *)malloc(size);
    r_max = (int *)malloc(size);
    r_need = (int *)malloc(size);
	for (p = 0; p < num_processes; p++){
		p_seen[p] = 0;
		p_sequence[p] = -1;
	}
	/* Fourth line in file is the resources available vector */
	read = getline(&line, &len, file);
	store_vector(r_avail,line);
	/* Read in the maximum reource needed matrix from file */
	for (p = 0; p < num_processes; p++){
		read = getline(&line, &len, file);
		tokens = strtok(line,",");
		while (tokens != NULL){
			r_max[index++] = atoi(tokens);
			tokens = strtok(NULL,",");
		}
	}
	/* Read in the reource allocation matrix from file */
	/* Calculate values for the need matrix */
    index = 0;
	for (p = 0; p < num_processes; p++){
		read = getline(&line, &len, file);
		tokens = strtok(line,",");
		while (tokens != NULL){
			r_alloc[index] = atoi(tokens);
			r_need[index] = r_max[index] - r_alloc[index];
			tokens = strtok(NULL,",");
			index++;
		}
	}
	fclose(file);
	if (line)
		free(line);
    hipEventRecord(start,0);
    t = clock();
    /* Allocate memory on the GPU */
    hipMalloc((void **)&gpu_r_avail, num_resources * sizeof(int));
    hipMalloc((void **)&gpu_r_alloc, size);
    hipMalloc((void **)&gpu_r_need, size);
    hipMalloc((void **)&gpu_r_out, num_processes * sizeof(int));
    hipMalloc((void **)&gpu_b_map, num_processes * sizeof(int));
    /* Copy alloc and need to GPU */
    hipMemcpy(gpu_r_avail, r_avail, num_resources * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(gpu_r_alloc, r_alloc, size, hipMemcpyHostToDevice);
    hipMemcpy(gpu_r_need, r_need, size, hipMemcpyHostToDevice);
    /* Launch kernel */
    int flag,counter;
    hipHostMalloc((void **)&r_out, num_processes * sizeof(int), hipHostMallocDefault);
    counter = 0;
    int blocks = num_processes / 100;
    int NUM_BLOCKS = num_processes;
    init_map<<<blocks,100>>>(gpu_b_map);
    while(counter < num_processes){
        //t = clock();
        set_zero<<<blocks,100>>>(gpu_r_out);
        //t = clock() - t;
        //printf("zero time: %f\n", ((double)t)/CLOCKS_PER_SEC);

        //t = clock();
        m_r_check<<<NUM_BLOCKS,THREADS_PER_BLOCK>>>(gpu_r_avail,gpu_r_need,gpu_r_out,gpu_b_map);
        //t = clock() - t;
        //printf("m_r check time: %f\n", ((double)t)/CLOCKS_PER_SEC);

        //t = clock();
        hipMemcpy(r_out, gpu_r_out, num_processes * sizeof(int), hipMemcpyDeviceToHost);
        //t = clock() - t;
        //printf("xfer time: %f\n", ((double)t)/CLOCKS_PER_SEC);

        flag = 0;
        for (p = 0; p < num_processes; p++){
            //printf("%d\n", r_out[p]);
            if (r_out[p] >= 0 && !p_seen[p]){
                //printf("%d ", p);
                add_r<<<1,THREADS_PER_BLOCK>>>(gpu_r_avail,gpu_r_alloc,p);
                update_map<<<blocks,100>>>(gpu_b_map,p);
                NUM_BLOCKS--;
                p_seen[p] = 1;
                p_sequence[counter++] = p;
                flag = 1;
                //break;
            }
        }
                //printf("\n");
        if (!flag){
                printf("DENIED");
                break;
        }
    }
    hipEventRecord(stop,0);
    hipEventSynchronize(stop);
    hipEventElapsedTime(&eTime, start, stop);
    hipEventDestroy(start);
    hipEventDestroy(stop);
    //printf("%s\n", cudaGetErrorString(cudaGetLastError()));
    /* Free GPU memory */
    hipFree(gpu_r_avail);
    hipFree(gpu_r_alloc);
    hipFree(gpu_r_need);
    hipFree(gpu_r_out);
    hipHostFree(r_out);
    t = clock() - t;

    if (argc == 3 && strcmp(argv[2],"-d") == 0){
        for (r = 0; r < num_processes; r++){
            printf("%d ", p_sequence[r]);
        }
        printf("\n\n");
    }

    printf("%d %f\n", num_processes, eTime/1000.0);
    //printf("%d %f\n", num_processes, ((double)t)/CLOCKS_PER_SEC);

    free(r_alloc);
    free(r_max);
    free(r_need);
	exit(EXIT_SUCCESS);
}
